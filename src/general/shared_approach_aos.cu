#include "hip/hip_runtime.h"
#include "../inc/parser.h"
#include "../inc/helper.cuh"
#include <errno.h>
#include <math.h>

__global__ void shared_approach_aos(Tuple* R, int rSize, Tuple* S, int sSize, int portionSize, Result* outputResults, int* outputCounter)
{
	extern __shared__ int shared[];
	int* sA = shared; // S.a
	int* sX = (int*)&sA[portionSize]; // S.x

	int rIndex = blockIdx.x * blockDim.x + threadIdx.x;

	//	//(rSize /blockDim.x) * blockDim.x + portionSize
	int threadID = threadIdx.x; //local thread id, i.e. within block

	int rA = R[rIndex].a; // store R.a to registers
	int sIndex = 0;

	while (sIndex < sSize) // iterate portions of the second array
	{
		if (threadID < portionSize)
		{
			sA[threadID] = S[sIndex + threadID].a;
			sX[threadID] = S[sIndex + threadID].x;
		}
		__syncthreads();

		//for (int i = 0; i < portionSize; ++i)
		for (int i = 0; i < (sSize - sIndex > portionSize ? portionSize : sSize - sIndex); ++i)
		{
			if (rA > sA[i] && rIndex < rSize)
			{
				outputResults[atomicAdd(outputCounter, 1)] = { rA, sA[i], sX[i] };
			}
		}
		sIndex += portionSize;

		__syncthreads();
	}
}

TimePair callKernel(Tuple* R, LLONG rSize, Tuple* S, LLONG sSize, int blockSize, int portionSize, int deviceMemorySize)
{
	float executionTime = 0.0;
	float millis = 0.0;

	hipEvent_t start, stop;
	checkErrors(hipEventCreate(&start));
	checkErrors(hipEventCreate(&stop));

	hipEvent_t* intermediate_start;
	hipEvent_t* intermediate_stop;

	Tuple* deviceR;
	Tuple* deviceS;

	// allocate memory for relations
	checkErrors(hipMalloc((void**)&(deviceR), sizeof(Tuple) * rSize));
	checkErrors(hipMalloc((void**)&(deviceS), sizeof(Tuple) * sSize));

	// copy relations to gpu
	checkErrors(hipMemcpy(deviceR, R, sizeof(Tuple) * rSize, hipMemcpyHostToDevice));
	checkErrors(hipMemcpy(deviceS, S, sizeof(Tuple) * sSize, hipMemcpyHostToDevice));

	Result* hostOutputResults;
	Result* deviceOutputResults;

	int* outputCounter;

	LLONG maxSpace = 0;
	LLONG md = deviceMemorySize * 1024LL * 1024LL * 1024LL;
	int t = 0;
	int side = 0;

	LLONG rIndex;
	LLONG sIndex;

	int rPortion;
	int sPortion;

	int tempOffset = 0;
	int offset = 0;
	int rectangles = 0;
	int count = 0;

	checkErrors(hipMalloc((void**)&(outputCounter), sizeof(int)));

	dim3 threadBlock(blockSize);
	dim3 grid((rSize / threadBlock.x) + 1);

	maxSpace = rSize * sSize * sizeof(Result);

	t = (maxSpace / md) + 1;

	printf("Md: %d, ||O||max: %llu, t: %d\n", deviceMemorySize, (maxSpace / 1024LL / 1024LL / 1024LL),t);
	printf("Grid: %dx%d, Thread Block: %dx%d\n", grid.x, grid.y, threadBlock.x, threadBlock.y);

	printf("Max results per pass %llu / %d = %d\n", md, sizeof(Result), (int)(md / (LLONG)sizeof(Result)));

	// allocate memory for device result
	checkErrors(hipMalloc((void**)&(deviceOutputResults), sizeof(Result) * (int)(md / (LLONG)sizeof(Result))));

	// allocate 10gb -> 894784853 host memory to store results from device (13gb -> 1163220310)
	hostOutputResults = (Result*) malloc(sizeof(Result) * 1163220310);

	checkErrors(hipEventRecord(start, 0));

	if(t == 1) 		// call kernel once and finish
	{
		shared_approach_aos <<<grid, threadBlock, portionSize * sizeof(int) + portionSize * sizeof(int), 0>>>(deviceR, rSize, deviceS, sSize, portionSize, deviceOutputResults, outputCounter);
		checkErrors(hipEventRecord(stop, 0));
		checkErrors(hipMemcpyAsync(&tempOffset, outputCounter, sizeof(int), hipMemcpyDeviceToHost, 0));
		checkErrors(hipMemcpyAsync(hostOutputResults, deviceOutputResults, tempOffset * sizeof(Result), hipMemcpyDeviceToHost, 0));
		checkErrors(hipEventElapsedTime(&executionTime, start, stop));
		offset += tempOffset;
	}
	else // call kernel multiple times
	{

		side = (int)sqrt((double)((rSize * sSize) / t));

		if((rSize % side == 0) && (sSize % side == 0) || ((rSize / t <= sSize ) && (sSize <= rSize)))
		{
			rPortion = side;
			sPortion = side;
		}
		else
		{
			rPortion = (int) rSize / t;
			sPortion = (int) sSize;
		}
		rectangles = getActualNumberOfRectangles(rPortion, sPortion, rSize, sSize);

		printf("Number of overall rectangles: %d\n", rectangles);

		intermediate_start = (hipEvent_t*)malloc(sizeof(hipEvent_t) * rectangles);
		intermediate_stop  = (hipEvent_t*)malloc(sizeof(hipEvent_t) * rectangles);

		for(int i = 0; i < rectangles; i++)
		{
			checkErrors(hipEventCreate(&intermediate_start[i]));
			checkErrors(hipEventCreate(&intermediate_stop[i]));
		}

		int zero = 0;
		rIndex = 0;
		while(rIndex < rSize)
		{
			sIndex = 0;
			while(sIndex < sSize)
			{
				if(rPortion > rSize - rIndex) rPortion = rSize - rIndex;
				if(sPortion > sSize - sIndex) sPortion = sSize - sIndex;

				checkErrors(hipEventRecord(intermediate_start[count], 0));
				shared_approach_aos <<<grid, threadBlock, portionSize * sizeof(int) + portionSize * sizeof(int), 0>>>(deviceR + rIndex, rPortion, deviceS + sIndex, sPortion, portionSize, deviceOutputResults, outputCounter);
				checkErrors(hipEventRecord(intermediate_stop[count], 0));

				checkErrors(hipMemcpyAsync(&tempOffset, outputCounter, sizeof(int), hipMemcpyDeviceToHost, 0));
				checkErrors(hipMemcpyAsync(outputCounter, &zero, sizeof(int), hipMemcpyHostToDevice, 0));
				checkErrors(hipMemcpyAsync(count == 0 ? hostOutputResults : hostOutputResults + offset, deviceOutputResults, tempOffset * sizeof(Result), hipMemcpyDeviceToHost, 0));

				offset += tempOffset;
				count++;
				sIndex += sPortion;
			}
			rIndex += rPortion;
		}

	}


	checkErrors(hipEventRecord(stop, 0));

	checkErrors(hipPeekAtLastError());
	checkErrors(hipDeviceSynchronize());

	checkErrors(hipEventSynchronize(stop));
	checkErrors(hipEventElapsedTime(&millis, start, stop));

	for(int i = 0; i < rectangles; i++)
	{
		float temp = 0.0;
		checkErrors(hipEventElapsedTime(&temp, intermediate_start[i], intermediate_stop[i]));
		executionTime += temp;
	}

	checkErrors(hipEventDestroy(start));
	checkErrors(hipEventDestroy(stop));

	for(int i = 0; i < rectangles; i++)
	{
		checkErrors(hipEventDestroy(intermediate_start[i]));
		checkErrors(hipEventDestroy(intermediate_stop[i]));
	}


	checkErrors(hipFree(deviceR));
	checkErrors(hipFree(deviceS));

	checkErrors(hipFree(deviceOutputResults));
	checkErrors(hipDeviceReset());

/*
	int res = assertResultsGenralAoS(R, rSize, S, sSize, hostOutputResults, offset);
	if(res == 0)
		printf("Success!\n");
	else
		printf("Fail :/\n");*/

	free(hostOutputResults);

	return {executionTime, millis};
}





int main(int argc, char** argv)
{
	char* rPath;
	LLONG rSize;

	char* sPath;
	LLONG sSize;

	int blockSize;
	int repeats;
	int deviceMemory;
	int portionSize;

	char* eptr;
	// read input arguments
	if (argc != 9)
	{
		printf("Not enough arguments\n---------------\n");
		printf("1st:\t R path\n");
		printf("2nd:\t |R| (R size)\n");
		printf("3rd:\t S path\n");
		printf("4th:\t |S| (S size)\n");
		printf("5th:\t Thread block size (max 1024)\n");
		printf("6th:\t Portion size (max 32)\n");
		printf("7th:\t Available Device Memory (in GB)\n");
		printf("8th:\t Number of repeats\n");
		return 1;
	}

	rPath = argv[1];
	sPath = argv[3];

	rSize = strtoll(argv[2], &eptr, 10);
	sSize = strtoll(argv[4], &eptr, 10);

	blockSize = strtol(argv[5], &eptr, 10);
	portionSize = strtol(argv[6], &eptr, 10);
	deviceMemory = strtol(argv[7], &eptr, 10);
	repeats = strtol(argv[8], &eptr, 10);

	if(rSize == 0 || sSize == 0 || blockSize == 0 || repeats == 0 || deviceMemory == 0 || portionSize == 0)
	{
		printf("Wrong input arguments (error: %d)", errno);
		return 1;
	}

	// allocate memory
	Tuple* R;
	Tuple* S;

	R = (Tuple*)malloc(sizeof(Relation) * rSize);
	S = (Tuple*)malloc(sizeof(Relation) * sSize);

	readRelationAoS(rPath, R);
	readRelationAoS(sPath, S);

	printf("Shared Memory Approach (AoS)\n");

	// call kernel multiple times
	TimePair* pairs = (TimePair*)malloc(sizeof(TimePair) * repeats);
	float executionTimeAggregate = 0.0;
	float overallTimeAggregate = 0.0;

	for(int i = 0; i < repeats; ++i)
	{
		pairs[i] = callKernel(R, rSize, S, sSize, blockSize, portionSize, deviceMemory);
		executionTimeAggregate += pairs[i].executionTime;
		overallTimeAggregate += pairs[i].overallTime;
	}

	// calculate and print average time
	printf("-----------------\n");
	printf("Execution time (GPU): %f\n", (executionTimeAggregate / (float) repeats));
	printf("Transfer & overhead time: %f\n", (overallTimeAggregate - executionTimeAggregate) / (float) repeats);
	printf("-----------------\n");
	printf("Overall Execution time: %f\n", (overallTimeAggregate / (float) repeats));

	free(R);
	free(S);
	free(pairs);

	return 0;
}
