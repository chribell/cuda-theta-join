#include "hip/hip_runtime.h"
#include "../inc/parser.h"
#include "../inc/helper.cuh"
#include <errno.h>
#include <conio.h>
#include <assert.h>
#include <windows.h>

template <int blockSize>
__global__ void shared_memory_approach_soa(int* Ra, LLONG rSize, int* Sa, int* Sx, LLONG sSize, LLONG* partialSums)
{
	extern __shared__ float sdata[];

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int blockID = gridDim.x * blockIdx.y + blockIdx.x;
	int threadID = (threadIdx.y * blockDim.x) + threadIdx.x; //local thread id, i.e. within block

	if(row > rSize) return;

	int rA = Ra[row];
	LLONG partialSum = 0;

	while (row < rSize && col < sSize)
	{
		if (rA > Sa[col])
		{
			partialSum += Sx[col];
		}
		col += blockDim.y;
	}

	sdata[threadID] = partialSum;
	__syncthreads();

	if ((blockSize >= 1024) && (threadID < 512))
	{
		sdata[threadID] = partialSum = partialSum + sdata[threadID + 512];
	}

	__syncthreads();

	if ((blockSize >= 512) && (threadID < 256))
	{
		sdata[threadID] = partialSum = partialSum + sdata[threadID + 256];
	}

	__syncthreads();

	if ((blockSize >= 256) && (threadID < 128))
	{
		sdata[threadID] = partialSum = partialSum + sdata[threadID + 128];
	}

	__syncthreads();

	if ((blockSize >= 128) && (threadID < 64))
	{
		sdata[threadID] = partialSum = partialSum + sdata[threadID + 64];
	}

	__syncthreads();

	if (threadID < 32)
	{
		// Fetch final intermediate sum from 2nd warp
		if (blockSize >= 64) partialSum += sdata[threadID + 32];
		// Reduce final warp using shuffle
		for (int offset = warpSize / 2; offset > 0; offset /= 2)
		{
			partialSum += __shfl_down(partialSum, offset);
		}
	}

	if (threadID == 0) partialSums[blockID] = partialSum;
}

float callKernel(Relation* R, LLONG rSize, Relation* S, LLONG sSize, dim3 threadBlock)
{
	float millis = 0.0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	unsigned int* deviceRindex;
	int* deviceRa;
	int* deviceRx;

	unsigned int* deviceSindex;
	int* deviceSa;
	int* deviceSx;

	LLONG* hostPartialSums;
	LLONG* devicePartialSums;

	// allocate memory for R relation
	checkErrors(hipMalloc((void**)&(deviceRindex), sizeof(unsigned int) * rSize));
	checkErrors(hipMalloc((void**)&(deviceRa), sizeof(int) * rSize));
	checkErrors(hipMalloc((void**)&(deviceRx), sizeof(int) * rSize));

	// allocate memory for S relation
	checkErrors(hipMalloc((void**)&(deviceSindex), sizeof(unsigned int) * sSize));
	checkErrors(hipMalloc((void**)&(deviceSa), sizeof(int) * sSize));
	checkErrors(hipMalloc((void**)&(deviceSx), sizeof(int) * sSize));

	// copy relations to gpu
	checkErrors(hipMemcpy(deviceRindex, R->index, sizeof(unsigned int) * rSize, hipMemcpyHostToDevice));
	checkErrors(hipMemcpy(deviceRa, R->a, sizeof(int) * rSize, hipMemcpyHostToDevice));
	checkErrors(hipMemcpy(deviceRx, R->x, sizeof(int) * rSize, hipMemcpyHostToDevice));

	checkErrors(hipMemcpy(deviceSindex, S->index, sizeof(unsigned int) * sSize, hipMemcpyHostToDevice));
	checkErrors(hipMemcpy(deviceSa, S->a, sizeof(int) * sSize, hipMemcpyHostToDevice));
	checkErrors(hipMemcpy(deviceSx, S->x, sizeof(int) * sSize, hipMemcpyHostToDevice));


	dim3 grid((rSize / threadBlock.x) + 1);

	// allocate memory for partialSums
	checkErrors(hipMalloc((void**)&(devicePartialSums), sizeof(LLONG) * grid.x));

	hipEventRecord(start);
	shared_memory_approach_soa<1024> <<<grid, threadBlock, 1024 * sizeof(LLONG)>>>(deviceRa, rSize, deviceSa, deviceSx, sSize, devicePartialSums);
	hipEventRecord(stop);

	checkErrors(hipPeekAtLastError());
	checkErrors(hipDeviceSynchronize());


	hostPartialSums = (LLONG*)malloc(sizeof(LLONG) * grid.x);
	checkErrors(hipMemcpy(hostPartialSums, devicePartialSums, sizeof(LLONG) * grid.x, hipMemcpyDeviceToHost));


	LLONG sum = 0;
	for(int i = 0; i < grid.x; ++i)
	{
		sum += hostPartialSums[i];
	}

	printf("Sum: %llu\n", sum);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&millis, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(deviceRindex);
	hipFree(deviceRa);
	hipFree(deviceRx);

	hipFree(deviceSindex);
	hipFree(deviceSa);
	hipFree(deviceSx);
	hipFree(devicePartialSums);
	hipDeviceReset();

	free(hostPartialSums);

	return millis;
}





int main(int argc, char** argv)
{
	char* rPath;
	LLONG rSize;

	char* sPath;
	LLONG sSize;

	int blockSideX;
	int blockSideY;
	int repeats;

	char* eptr;
	// read input arguments
	if (argc != 8)
	{
		printf("Not enough arguments\n---------------\n");
		printf("1st:\t R path\n");
		printf("2nd:\t |R| (R size)\n");
		printf("3rd:\t S path\n");
		printf("4th:\t |S| (S size)\n");
		printf("5th:\t Thread block side x \n");
		printf("6th:\t Thread block side y \n");
		printf("7th:\t Number of repeats\n");
		return 1;
	}

	rPath = argv[1];
	sPath = argv[3];

	rSize = strtoll(argv[2], &eptr, 10);
	sSize = strtoll(argv[4], &eptr, 10);

	blockSideX = strtol(argv[5], &eptr, 10);
	blockSideY = strtol(argv[6], &eptr, 10);
	repeats = strtol(argv[7], &eptr, 10);

	if(rSize == 0 || sSize == 0 || blockSideX == 0 || blockSideY == 0 || repeats == 0)
	{
		printf("Wrong input arguments (error: %d)", errno);
		return 1;
	}

	// allocate memory
	Relation* R;
	Relation* S;

	R = (Relation*)malloc(sizeof(Relation));
	S = (Relation*)malloc(sizeof(Relation));

	readRelationSoA(rPath, R, rSize);
	readRelationSoA(sPath, S, sSize);

	printf("Shared Memory Approach (SoA)\n");

	// call kernel multiple times
	float time_aggregate = 0.0;
	dim3 threadBlock(blockSideX, blockSideY);
	for(int i = 0; i < repeats; ++i)
	{
		time_aggregate += callKernel(R, rSize, S, sSize, threadBlock);
	}

	// calculate and print average time
	printf("Execution time: %f\n", (time_aggregate / (float) repeats));

	free(R);
	free(S);

	return 0;
}
